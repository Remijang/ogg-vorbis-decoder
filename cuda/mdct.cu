#include "hip/hip_runtime.h"
#include "mdct.h"
#include <math.h>
#include <vector>

__global__ void easy_IMDCT_kernel (double *X, double *y, double *window, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (idx < n) {
		y[idx] = 0.0;
		for(int m = 0; m < n / 2; ++m) {
			y[idx] += X[m] * cos(M_PI / 2 / n * (2 * idx + 1 + n / 2) * (2 * m + 1));
		}
		y[idx] *= window[idx];
		idx += stride;
	}
}
void IMDCT_gpu (double *X, double *y, double *window, int n) {
	static double *X_d, *y_d, *window_d;
	static int ok = 0;
	if (!ok) {
		hipMalloc(&X_d, sizeof(double) * 8192);
		hipMalloc(&y_d, sizeof(double) * 8192);
		hipMalloc(&window_d, sizeof(double) * 8192);
		ok++;
	}
	hipMemcpy(X_d, X, sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(window_d, window, sizeof(double) * n, hipMemcpyHostToDevice);
	int sm_count = n / 256;
	easy_IMDCT_kernel<<<sm_count, 256>>>(X_d, y_d, window_d, n);
	hipDeviceSynchronize();
	hipMemcpy(y, y_d, sizeof(double) * n, hipMemcpyDeviceToHost);
}
